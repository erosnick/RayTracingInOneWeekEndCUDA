#include "hip/hip_runtime.h"
﻿
#include "main.h"
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"
#include "Utils.h"
#include "GPUTimer.h"
#include "Sphere.h"
#include <cstdio>

template<typename T>
T* createObjectPtr() {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T*)));
    return object;
}

template<typename T>
T* createObjectArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T) * numObjects));
    return object;
}

template<typename T>
T* createObjectPtrArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T*) * numObjects));
    return object;
}

template<typename T>
void deleteObject(T* object) {
    gpuErrorCheck(hipFree(object));
}

template<typename T>
CUDA_GLOBAL void deleteDeviceObject(T** object) {
    delete (*object);
}

constexpr auto SPHERES = 5;
CUDA_CONSTANT Sphere constantSpheres[SPHERES];

CUDA_DEVICE bool hit(const Ray& ray, Float tMin, Float tMax, HitResult& hitResult, Sphere* spheres) {
    HitResult tempHitResult;
    bool bHitAnything = false;
    Float closestSoFar = tMax;
    //for (auto& sphere : constantSpheres) {
    for (auto i = 0; i < SPHERES; i++){
        auto sphere = spheres[i];
        if (!sphere.bShading) {
            continue;
        }
        if (sphere.hit(ray, tMin, closestSoFar, tempHitResult)) {
            bHitAnything = true;
            closestSoFar = tempHitResult.t;
            hitResult = tempHitResult;
        }
    }

    return bHitAnything;
}

CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState* randState, Sphere* spheres) {
    Ray currentRay = ray;
    auto currentAttenuation = make_float3(1.0f, 1.0f, 1.0f);
    for (auto i = 0; i < 50; i++) {
        HitResult hitResult;
        // Smaller tMin will has a impact on performance
        if (hit(currentRay, Math::epsilon, Math::infinity, hitResult, spheres)) {
            Float3 attenuation;
            Ray scattered;
            if (hitResult.material->scatter(currentRay, hitResult, attenuation, scattered, randState)) {
                currentAttenuation *= attenuation;
                currentRay = scattered;
            }
            else {
                return make_float3(0.0f, 0.0f, 0.0f);
            }
        }
        else {
            auto unitDirection = normalize(currentRay.direction);
            auto t = 0.5f * (unitDirection.y + 1.0f);
            auto background = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
            return currentAttenuation * background;
        }
    }

    // exceeded recursion
    return make_float3(0.0f, 0.0f, 0.0f);
}

//CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState* randState, Sphere* spheres, int32_t depth) {
//    if (depth == 0) {
//        // exceeded recursion
//        return make_float3(0.0f, 0.0f, 0.0f);
//    }
//    HitResult hitResult;
//    // Smaller tMin will has a impact on performance
//    if (hit(ray, Math::epsilon, Math::infinity, hitResult, spheres)) {
//        Float3 attenuation;
//        Ray rayScattered;
//        if (hitResult.material->scatter(ray, hitResult, attenuation, rayScattered, randState)) {
//            return attenuation * rayColor(rayScattered, randState, spheres, depth - 1);
//        }
//        else {
//            return make_float3(0.0f, 0.0f, 0.0f);
//        }
//    }
//
//    auto unitDirection = normalize(ray.direction);
//    auto t = 0.5f * (unitDirection.y + 1.0f);
//    auto background = lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
//    return background;
//}

CUDA_GLOBAL void renderInit(int32_t width, int32_t height, hiprandState* randState) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto index = y * width + x;

    if (index < (width * height)) {
        //Each thread gets same seed, a different sequence number, no offset
        hiprand_init(1984, index, 0, &randState[index]);
    }
}

//CUDA_GLOBAL void render(Canvas canvas, Camera camera, hiprandState* randStates, Sphere* spheres) {
//    auto x = threadIdx.x + blockDim.x * blockIdx.x;
//    auto y = threadIdx.y + blockDim.y * blockIdx.y;
//    auto width = canvas.getWidth();
//    auto height = canvas.getHeight();
//    constexpr auto samplesPerPixel = 1;
//    constexpr auto maxDepth = 5;
//    auto index = y * width + x;
//
//    if (index < (width * height)) {
//        auto color = make_float3(0.0f, 0.0f, 0.0f);
//        auto localRandState = randStates[index];
//        for (auto i = 0; i < samplesPerPixel; i++) {
//
//            auto rx = hiprand_uniform(&localRandState);
//            auto ry = hiprand_uniform(&localRandState);
//
//            auto dx = Float(x + rx) / (width - 1);
//            auto dy = Float(y + ry) / (height - 1);
//
//            auto ray = camera.getRay(dx, dy);
//            color += rayColor(ray, &localRandState, spheres);
//        }
//        // Very important!!!
//        randStates[index] = localRandState;
//        canvas.writePixel(index, color / samplesPerPixel);
//    }
//}

CUDA_GLOBAL void render(Canvas* canvas, Camera* camera, hiprandState* randStates, Sphere* spheres) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = canvas->getWidth();
    auto height = canvas->getHeight();
    constexpr auto samplesPerPixel = 1;
    constexpr auto maxDepth = 5;
    auto index = y * width + x;

    if (index < (width * height)) {
        auto color = make_float3(0.0f, 0.0f, 0.0f);
        auto localRandState = randStates[index];
        for (auto i = 0; i < samplesPerPixel; i++) {

            auto rx = hiprand_uniform(&localRandState);
            auto ry = hiprand_uniform(&localRandState);

            auto dx = Float(x + rx) / (width - 1);
            auto dy = Float(y + ry) / (height - 1);

            auto ray = camera->getRay(dx, dy);
            color += rayColor(ray, &localRandState, spheres);
        }
        // Very important!!!
        randStates[index] = localRandState;
        //canvas->writePixel(index, color / samplesPerPixel);
        canvas->accumulatePixel(index, color);
    }
}

CUDA_GLOBAL void createLambertianMaterial(Material** material, Float3 albedo, Float absorb = 1.0f) {
    (*material) = new Lambertian(albedo, absorb);
}

CUDA_GLOBAL void createMetalMaterial(Material** material, Float3 albedo, Float fuzz = 1.0f) {
    (*material) = new Metal(albedo, fuzz);
}

CUDA_GLOBAL void createDieletricMaterial(Material** material, Float indexOfRefraction = 1.5f) {
    (*material) = new Dieletric(indexOfRefraction);
}

CUDA_GLOBAL void clearBackBuffers(Canvas* canvas) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = canvas->getWidth();
    auto height = canvas->getHeight();

    auto index = y * width + x;

    if (index < (width * height)) {
        canvas->clearPixel(index);
    }
}

#define RESOLUTION 0

#if RESOLUTION == 0
int32_t width = 512;
int32_t height = 288;
#elif RESOLUTION == 1
int32_t width = 1024;
int32_t height = 576;
#elif RESOLUTION == 2
int32_t width = 1280;
int32_t height = 720;
#elif RESOLUTION == 3
int32_t width = 1920;
int32_t height = 1080;
#elif RESOLUTION == 4
int32_t width = 64;
int32_t height = 36;
#endif

int32_t sampleCount = 0;

Canvas* canvas = nullptr;
Camera* camera = nullptr;
Sphere* spheres = nullptr;
Material** materials[SPHERES];
hiprandState* randStates = nullptr;
std::shared_ptr<ImageData> imageData = nullptr;

dim3 blockSize(32, 32);
dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
              (height + blockSize.y - 1) / blockSize.y);

void initialize(int32_t width, int32_t height) {
    //Canvas canvas(width, height);
    Utils::reportGPUUsageInfo();
    canvas = createObjectPtr<Canvas>();
    canvas->initialize(width, height);
    Utils::reportGPUUsageInfo();
    //Camera camera(make_float3(-2.0f, 2.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 20.0f);
    camera = createObjectPtr<Camera>();
    camera->initialize(make_float3(-2.0f, 2.0f, 1.0f), make_float3(0.0f, 0.0f, -1.0f), make_float3(0.0f, 1.0f, 0.0f), Float(width) / height, 20.0f);

    spheres = createObjectArray<Sphere>(SPHERES);

    for (auto& material : materials) {
        material = createObjectPtr<Material*>();
    }

    createDieletricMaterial<<<1, 1>>>(materials[0], 1.5f);
    createDieletricMaterial<<<1, 1>>>(materials[1], 1.5f);
    createLambertianMaterial<<<1, 1>>>(materials[2], make_float3(0.1f, 0.2f, 0.5f));
    //createDieletricMaterial<<<1, 1>>>(materials[3], 1.5f);
    createMetalMaterial<<<1, 1>>>(materials[3], make_float3(0.8f, 0.6f, 0.2f), 0.0f);
    createLambertianMaterial<<<1, 1>>>(materials[4], make_float3(0.8f, 0.8f, 0.0f));
    gpuErrorCheck(hipDeviceSynchronize());

    spheres[0] = { { -1.0f, 0.0f, -1.0f},   0.5f, *(materials[0]), true };
    spheres[1] = { { -1.0f, 0.0f, -1.0f }, -0.4f, *(materials[1]), false };
    spheres[2] = { {  0.0f, 0.0f, -1.0f },  0.5f, *(materials[2]), true };
    spheres[3] = { {  1.0f, 0.0f, -1.0f },  0.5f, *(materials[3]), true };
    spheres[4] = { {  0.0f, -100.5f, -1.0f }, 100.0f, *(materials[4]), true };

    auto pixelCount = width * height;
    randStates = createObjectArray<hiprandState>(pixelCount);

    renderInit<<<gridSize, blockSize>>>(width, height, randStates);
    gpuErrorCheck(hipDeviceSynchronize());

    imageData = std::make_shared<ImageData>();

    imageData->width = width;
    imageData->height = height;
    imageData->channels = 3;
    imageData->size = pixelCount * 3;
}   

void clearBackBuffers() {
    clearBackBuffers<<<gridSize, blockSize>>>(canvas);
    gpuErrorCheck(hipDeviceSynchronize());
    canvas->resetSampleCount();
}

void pathTracing() {
    if (camera->isDirty()) {
        clearBackBuffers();
        camera->resetDiryFlag();
    }

    canvas->incrementSampleCount();
    render<<<gridSize, blockSize>>>(canvas, camera, randStates, spheres);
    gpuErrorCheck(hipDeviceSynchronize());

    imageData->data = canvas->getPixelBuffer();
}

void cleanup() {
    deleteObject(randStates);

    for (auto i = 0; i < SPHERES; i++) {
        deleteDeviceObject<<<1, 1>>>(materials[i]);
        gpuErrorCheck(hipDeviceSynchronize());
        gpuErrorCheck(hipFree(materials[i]));
}

    deleteObject(spheres);

    deleteObject(camera);
    canvas->uninitialize();
    deleteObject(canvas);
}

#ifndef GPU_REALTIME
int main() {
    //gpuErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 8192));

    initialize(width, height);

    //gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constantSpheres), spheres, sizeof(Sphere) * SPHERES));
    
    GPUTimer timer("Rendering start...");
    pathTracing();
    timer.stop("Rendering elapsed time");

    canvas->writeToPNG("render.png");
    Utils::openImage(L"render.png");

    cleanup();

    return 0;
}
#endif // !GPU_REALTIME