#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Utils.h"
#include "Canvas.h"
#include "GPUTimer.h"
#include "Camera.h"
#include "Sphere.h"
#include <cstdio>

template<typename T>
T* createObject() {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T*)));
    return object;
}

template<typename T>
T* createObjectArray(int32_t numObjects) {
    T* object = nullptr;
    gpuErrorCheck(hipMallocManaged(&object, sizeof(T) * numObjects));
    return object;
}

template<typename T>
void deleteObject(T* object) {
    gpuErrorCheck(hipFree(object));
}

constexpr auto SPHERES = 2;
CUDA_CONSTANT Sphere constantSpheres[SPHERES];

CUDA_DEVICE bool hit(const Ray& ray, Float tMin, Float tMax, HitResult& hitResult) {
    HitResult tempHitResult;
    bool bHitAnything = false;
    Float closestSoFar = tMax;
    for (auto& sphere : constantSpheres) {
        if (sphere.hit(ray, tMin, closestSoFar, tempHitResult)) {
            bHitAnything = true;
            closestSoFar = tempHitResult.t;
            hitResult = tempHitResult;
        }
    }

    return bHitAnything;
}

CUDA_DEVICE Float3 rayColor(const Ray& ray, hiprandState& randState) {
    HitResult hitResult;
    if (hit(ray, Math::epsilon, Math::infinity, hitResult)) {
        return 0.5f * (hitResult.normal + 1.0f);
    }

    auto unitDirection = normalize(ray.direction);
    auto t = 0.5f * (unitDirection.y + 1.0f);
    return lerp(make_float3(1.0f, 1.0f, 1.0f), make_float3(0.5f, 0.7f, 1.0f), t);
}

CUDA_GLOBAL void renderInit(int32_t width, int32_t height, hiprandState* randState) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto index = y * width + x;

    if (index < (width * height)) {
        //Each thread gets same seed, a different sequence number, no offset
        hiprand_init(1984, index, 0, &randState[index]);
    }
}

CUDA_GLOBAL void render(Canvas canvas, Camera camera, hiprandState* randStates) {
    auto x = threadIdx.x + blockDim.x * blockIdx.x;
    auto y = threadIdx.y + blockDim.y * blockIdx.y;
    auto width = camera.getImageWidth();
    auto height = camera.getImageHeight();
    constexpr auto samplesPerPixel = 8;

    auto index = y * width + x;

    if (index < (width * height)) {
        auto color = make_float3(0.0f, 0.0f, 0.0f);
        auto localRandState = randStates[index];
        for (auto i = 0; i < samplesPerPixel; i++) {

            auto rx = hiprand_uniform(&localRandState);
            auto ry = hiprand_uniform(&localRandState);

            auto dx = Float(x + rx) / (width - 1);
            auto dy = Float(y + ry) / (height - 1);

            auto ray = camera.getRay(dx, dy);

            color += rayColor(ray, localRandState);
        }

        canvas.writePixel(index, color / samplesPerPixel);
    }
}

int main() {
    //gpuErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 8192));

    constexpr auto width = 1280;
    constexpr auto height = 720;
    constexpr auto pixelCount = width * height;

    Canvas canvas(width, height);
    //auto* canvas = createObject<Canvas>();
    //canvas->initialize(width, height);

    Camera camera(width, height);
    //auto* camera = createObject<Camera>();
    //camera->initialize(width, height);

    Sphere spheres[SPHERES];

    spheres[0].center = {0.0f, 0.0f, -1.0f};
    spheres[0].color = make_float3(1.0f, 0.0f, 0.0f);
    spheres[0].radius = 0.5f;

    spheres[1].center = { 0.0f, -100.5f, -1.0f };
    spheres[1].color = make_float3(1.0f, 0.0f, 0.0f);
    spheres[1].radius = 100.0f;

    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(constantSpheres), spheres, sizeof(Sphere) * SPHERES));

    auto* randStates = createObjectArray<hiprandState>(pixelCount);

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    renderInit<<<gridSize, blockSize>>>(width, height, randStates);
    gpuErrorCheck(hipDeviceSynchronize());

    GPUTimer timer("Rendering start...");

    render<<<gridSize, blockSize>>>(canvas, camera, randStates);
    gpuErrorCheck(hipDeviceSynchronize());

    timer.stop("Rendering elapsed time");

    canvas.writeToPNG("render.png");
    Utils::openImage(L"render.png");

    deleteObject(randStates);

    //deleteObject(camera);

    canvas.uninitialize();
    //deleteObject(canvas);

    return 0;
}
